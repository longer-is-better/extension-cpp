#include "hip/hip_runtime.h"
#include <ATen/Operators.h>
#include <torch/all.h>
#include <torch/library.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <ATen/cuda/HIPContext.h>

#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                    hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

namespace extension_cpp {


__device__ float warpMax(float val) {
    for (int offset = 16; offset > 0; offset >>= 1) 
        val = fmaxf(val, __shfl_down_sync(0xffffffff, val, offset));
    return val;
}

__device__ float warpSum(float val) {
    for (int offset = 16; offset > 0; offset >>= 1) 
        val += __shfl_down_sync(0xffffffff, val, offset);
    return val;
}

// one block on one SM
// 
__global__ void flashattention_kernel(
    uint Tr, uint Tc, uint Br, uint Bc, uint head_dim, uint element_size,
    const float* q, const float* k, const float* v, float* l, float* m, float* o
) {
    extern __shared__ float smem[];
    float* shared_q = smem;
    float* shared_k = shared_q + Br * head_dim * element_size;
    float* shared_v = shared_k + Bc * head_dim * element_size;
    float* shared_pv = shared_v + Bc * head_dim * element_size;
    float* shared_o = shared_pv + Bc * head_dim * element_size;
    float* shared_l = shared_o + Br * head_dim * element_size;
    float* shared_m = shared_l + Br * element_size;

    uint load_kv_blocknum = (head_dim + Br - 1) / Br;
    uint load_qo_blocknum = (head_dim + Bc - 1) / Bc;
    uint cal_pv_blocknum = load_qo_blocknum;

    for (int c = 0; c < Tc; c++) {
        // load k v
        for (int i = 0; i < load_kv_blocknum; i++) {
            if (i * Br + threadIdx.y >= head_dim) continue; 

            shared_k[
                i * Br * Bc + threadIdx.y * Bc + threadIdx.x
            ] = k[
                c * Bc * head_dim + threadIdx.x * head_dim + i * Br + threadIdx.y
            ];

            shared_v[
                threadIdx.x * head_dim + i * Br + threadIdx.y
            ] = v[
                c * Bc * head_dim + threadIdx.x * head_dim + i * Br + threadIdx.y
            ];
        }
        for (int r = 0; r < Tr; r++) {
            // load q o l m
            for (int i = 0; i < load_qo_blocknum; i++) {
                shared_q[
                    threadIdx.y * head_dim + Bc * i + threadIdx.x
                ] = q[
                    r * Br * head_dim + threadIdx.y * head_dim + Bc * i + threadIdx.x
                ];
                
                shared_o[
                    threadIdx.y * head_dim + Bc * i + threadIdx.x
                ] = o[
                    r * Br * head_dim + threadIdx.y * head_dim + Bc * i + threadIdx.x
                ];

                if (threadIdx.x == 0) {
                    shared_l[threadIdx.y] = l[r * Br + threadIdx.y];
                    shared_m[threadIdx.y] = m[r * Br + threadIdx.y];
                }
            }
            // calculate one cell of QKt
            float s = 0;
            for (int i = 0; i < head_dim; i++) {
                s = __fmaf_rn(shared_q[threadIdx.y * head_dim + i], shared_k[i * Bc + threadIdx.x], s);
            }
            // reduce max _m and broadcast https://zhuanlan.zhihu.com/p/669957986
            float _m = warpMax(s);
            _m = __shfl_sync(0xffffffff, _m, 0);
            // cal p l
            float _p = expf(s - _m);
            float _l = warpSum(_p);
            _l = __shfl_sync(0xffffffff, _l, 0);
            float _m_new = max(shared_m[threadIdx.y], _m);
            float scale_old = expf(shared_m[threadIdx.y] - _m_new);
            float scale_new = expf(_m - _m_new);
            float _l_new = scale_old * shared_l[threadIdx.y] + scale_new * _l;
            // calculate PV
            for (int head = 0; head < head_dim; head++) {
                atomicAdd(shared_pv + threadIdx.y * head_dim + head, shared_v[threadIdx.x * head_dim + head] * _p);
            }
            for (int i = 0; i < cal_pv_blocknum; i++) {
                o[r * Br * head_dim + threadIdx.y * head_dim + Bc * i + threadIdx.x] = (shared_l[threadIdx.y] * scale_old * shared_o[threadIdx.y * head_dim + cal_pv_blocknum * i + threadIdx.x] + scale_new * shared_pv[threadIdx.y * head_dim + cal_pv_blocknum * i + threadIdx.x]) / _l_new;
            }
            l[r * Br + threadIdx.y] = _l_new;
            m[r * Br + threadIdx.y] = _m_new;
        }
    }
}




at::Tensor flashattention_cuda(
    const at::Tensor& q, const at::Tensor& k, const at::Tensor& v
) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    uint M = deviceProp.sharedMemPerBlock;
    uint element_size = q.element_size();
    uint seq_len = q.size(0);
    uint head_dim = q.size(1);
    // uint Bc = M / (4 * head_dim * element_size);
    // uint Br = min(Bc, head_dim);
    uint Bc = 32;  //warpSize
    uint Br = (M - 2 * Bc * head_dim * element_size) / (3 * (head_dim + 1) * element_size);
    Br = min(Br, 32);  //warpSize
    uint Tr = (seq_len + Br - 1) / Br;
    uint Tc = (seq_len + Bc - 1) / Bc;
    at::Tensor q_contig = q.contiguous();
    at::Tensor k_contig = k.contiguous();
    at::Tensor v_contig = v.contiguous();
    at::Tensor l = at::zeros({seq_len}, torch::dtype(torch::kFloat32).device(torch::kCUDA));
    at::Tensor m = at::full(
        {seq_len},
        -std::numeric_limits<float>::infinity(),
        torch::dtype(torch::kFloat32).device(torch::kCUDA)
    );
    at::Tensor result = at::zeros({seq_len, head_dim}, torch::dtype(torch::kFloat32).device(torch::kCUDA));
    uint smem = ((2 * Bc + 3 * Br) * head_dim + 2 * Br) * element_size;
    flashattention_kernel<<<{1}, {Bc, Br}, smem, at::cuda::getCurrentCUDAStream()>>>(
        Tr, Tc, Br, Bc, head_dim, element_size,
        q_contig.data_ptr<float>(),
        k_contig.data_ptr<float>(),
        v_contig.data_ptr<float>(),
        l.data_ptr<float>(),
        m.data_ptr<float>(),
        result.data_ptr<float>()
    );
    std::cout << "start" << std::endl;
    CUDA_CHECK(hipGetLastError());
    std::cout << "wait" << std::endl;
    CUDA_CHECK(hipDeviceSynchronize());
    return result;
}

// Registers CUDA implementations for mymuladd, mymul, myadd_out
TORCH_LIBRARY_IMPL(extension_cpp, CUDA, m) {
    m.impl("flashattention", &flashattention_cuda);
  }
  
}
